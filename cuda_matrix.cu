
//  nvcc -o matrix_multiply matrix_multiply.cu



//Create a C++ project and include the necessary headers:
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>


//Define a function to allocate and initialize matrices:
void initializeMatrix(std::vector<float>& mat, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            mat[i * cols + j] = static_cast<float>(rand() % 100);
        }
    }
}


//Define the matrix multiplication kernel:
__global__ void matrixMultiplyKernel(const float* A, const float* B, float* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}


//Define the main function:
int main() {
    const int m = 4;  // Number of rows in A
    const int n = 3;  // Number of columns in A and rows in B
    const int k = 5;  // Number of columns in B

    // Allocate and initialize matrices A and B
    std::vector<float> hostA(m * n);
    std::vector<float> hostB(n * k);
    std::vector<float> hostC(m * k);
    initializeMatrix(hostA, m, n);
    initializeMatrix(hostB, n, k);

    // Allocate device memory for A, B, and C
    float *deviceA, *deviceB, *deviceC;
    hipMalloc(&deviceA, m * n * sizeof(float));
    hipMalloc(&deviceB, n * k * sizeof(float));
    hipMalloc(&deviceC, m * k * sizeof(float));

    // Copy matrices A and B from host to device
    hipMemcpy(deviceA, hostA.data(), m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB.data(), n * k * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions for the kernel
    dim3 gridDim(2, 2); // You can adjust these dimensions based on your GPU and matrix size
    dim3 blockDim(2, 2);

    // Launch the matrix multiplication kernel
    matrixMultiplyKernel<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC, m, n, k);
    hipDeviceSynchronize();

    // Copy the result matrix C from the device to host
    hipMemcpy(hostC.data(), deviceC, m * k * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    // Output the result
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < k; ++j) {
            std::cout << hostC[i * k + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}







